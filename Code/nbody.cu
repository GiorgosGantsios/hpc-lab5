#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
// #include <omp.h>

#define SOFTENING 1e-9f  /* Will guard against denormals */
#define tolerance 0.005// 2 decimals
/* Macro for error checking in CUDA calls */
#define CHECK_CUDA_ERROR(call)                                \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      fprintf(stderr, "CUDA error at %s:%d: %s\n",            \
              __FILE__, __LINE__, hipGetErrorString(err));   \
      goto cleanup;                                           \
    }                                                         \
  } while (0)
typedef struct { float x, y, z, vx, vy, vz; } Body;

void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

void bodyForce(Body *p, float dt, int n) {
  for (int i = 0; i < n; i++) { 
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int j = 0; j < n; j++) {
      float dx = p[j].x - p[i].x;
      float dy = p[j].y - p[i].y;
      float dz = p[j].z - p[i].z;
      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      float invDist = 1.0f / sqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }

    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
  }
}

int main(const int argc, const char** argv) {
  
  int nBodies = 30000;
  if (argc > 1) nBodies = atoi(argv[1]);

  const float dt = 0.01f; // time step
  const int nIters = 10;  // simulation iterations
  float *buf;
  Body *dbuf;
  Body *hbuf;
  Body *p;
  int bytes = nBodies*sizeof(Body);
  hipError_t e = hipSuccess;
  double avgTime = 0.0;
  double totalTime = 0.0;

  buf = (float*)malloc(bytes);
  if(!buf) goto cleanup;
  hbuf = (Body*)malloc(bytes);
  if(!hbuf) goto cleanup;
  CHECK_CUDA_ERROR(hipMalloc((void**)&dbuf, bytes));

  p = (Body*)buf;

  randomizeBodies(buf, 6*nBodies); // Init pos / vel data

  for (int iter = 1; iter <= nIters; iter++) {
    StartTimer();

    bodyForce(p, dt, nBodies); // compute interbody forces
    
    // #pragma omp parallel for
    for (int i = 0 ; i < nBodies; i++) { // integrate position
      p[i].x += p[i].vx*dt;
      p[i].y += p[i].vy*dt;
      p[i].z += p[i].vz*dt;
    }

    const double tElapsed = GetTimer() / 1000.0;
    if (iter > 1) { // First iter is warm up
      totalTime += tElapsed; 
    }
    printf("Iteration %d: %.3f seconds\n", iter, tElapsed);
  }
  avgTime = totalTime / (double)(nIters-1); 

  printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, 1e-9 * nBodies * nBodies / avgTime);
  
  CHECK_CUDA_ERROR(hipMemcpy(hbuf, dbuf, bytes, hipMemcpyDeviceToHost));
  
  // Chech similarity between CPU (buf) and GPU (hbuf)
  for (int i = 0; i < nBodies; i++) {
    // Compare positions and velocities between CPU and GPU
    if (fabs(p[i].x - hbuf[i].x) > tolerance ||
        fabs(p[i].y - hbuf[i].y) > tolerance ||
        fabs(p[i].z - hbuf[i].z) > tolerance ||
        fabs(p[i].vx - hbuf[i].vx) > tolerance ||
        fabs(p[i].vy - hbuf[i].vy) > tolerance ||
        fabs(p[i].vz - hbuf[i].vz) > tolerance) {
          printf("Difference bigger than tolerance in index: %d\n", i);
          goto cleanup;
    }
  }
  
  
  
  
  
  cleanup:
  if (buf) free(buf);
  if (dbuf) hipFree(dbuf);

  hipDeviceSynchronize();
  e = hipGetLastError();
  if(e!=hipSuccess){
    printf("ERROR: %s, FILE: %s, LINE: %d\n", hipGetErrorString(e), __FILE__, __LINE__);
  }
  else{
    printf("hipGetLastError() == hipSuccess!\n");
  }

  hipDeviceReset();
  
  return 0;
}
