#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
// #include <omp.h>

#define SOFTENING 1e-9f  /* Will guard against denormals */
#define tolerance 0.1// 2 decimals
/* Macro for error checking in CUDA calls */
#define CHECK_CUDA_ERROR(call)                                \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      fprintf(stderr, "CUDA error at %s:%d: %s\n",            \
              __FILE__, __LINE__, hipGetErrorString(err));   \
      goto cleanup;                                           \
    }                                                         \
  } while (0)
typedef struct { float x, y, z, vx, vy, vz; } Body;

void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

void bodyForce(Body *p, float dt, int n) {
  for (int i = 0; i < n; i++) { 
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int j = 0; j < n; j++) {
      float dx = p[j].x - p[i].x;
      float dy = p[j].y - p[i].y;
      float dz = p[j].z - p[i].z;
      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      float invDist = 1.0f / sqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }

    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
  }
}

__global__ void bodyForce_gpu(Body *p, float dt, int n) {
  float distSqr;
  float invDist;
  float invDist3;
  int j;//, i;
  float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  for (j = 0; j < n; j++) {
      float dx = p[j].x - p[index].x;
      float dy = p[j].y - p[index].y;
      float dz = p[j].z - p[index].z;
      distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      invDist = 1.0f / sqrtf(distSqr);
      invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
  }

  p[index].vx += dt*Fx; p[index].vy += dt*Fy; p[index].vz += dt*Fz;

__syncthreads();

  p[index].x += p[index].vx*dt;
  p[index].y += p[index].vy*dt;
  p[index].z += p[index].vz*dt;
}

int check_correctness(Body *hbuf, Body *p, int nBodies, int *diff_counter){
  int correct = 1; // flag to allow only 1 print
  for (int i = 0; i < nBodies; i++) {
    // Compare positions and velocities between CPU and GPU
    if (fabs(p[i].x - hbuf[i].x) > tolerance ||
        fabs(p[i].y - hbuf[i].y) > tolerance ||
        fabs(p[i].z - hbuf[i].z) > tolerance ||
        fabs(p[i].vx - hbuf[i].vx) > tolerance ||
        fabs(p[i].vy - hbuf[i].vy) > tolerance ||
        fabs(p[i].vz - hbuf[i].vz) > tolerance) {
        if(correct){
          correct = 0;
          printf("Difference exceeds tolerance at index: %d\n", i);
          printf("Position - CPU: (%.6f, %.6f, %.6f), GPU: (%.6f, %.6f, %.6f)\n",
                 p[i].x, p[i].y, p[i].z, hbuf[i].x, hbuf[i].y, hbuf[i].z);
          printf("Velocity - CPU: (%.6f, %.6f, %.6f), GPU: (%.6f, %.6f, %.6f)\n",
                 p[i].vx, p[i].vy, p[i].vz, hbuf[i].vx, hbuf[i].vy, hbuf[i].vz);
        }
        (*diff_counter)++;
    }
  }

  return correct;
}

int main(const int argc, const char** argv) {
  
  int nBodies = 30000;
  if (argc > 1) nBodies = atoi(argv[1]);

  const float dt = 0.01f; // time step
  const int nIters = 10;  // simulation iterations
  float *buf;
  Body *dbuf;
  Body *hbuf;
  Body *p, *buf_1iter;
  int bytes = nBodies*sizeof(Body);
  hipError_t e = hipSuccess;
  hipEvent_t startCuda, stopCuda;
  double avgTime = 0.0;
  double totalTime = 0.0;
  int extra_block;
  float msecBeforeCheck = 0.0f, msecAfterCheck = 0.0f;
  float msec = 0.0f;
  int diff_counter = 0;

  buf = (float*)malloc(bytes);
  if(!buf) goto cleanup;
  buf_1iter = (Body*)malloc(bytes);
  if(!buf_1iter) goto cleanup;
  hbuf = (Body*)malloc(bytes);
  if(!hbuf) goto cleanup;
  CHECK_CUDA_ERROR(hipMalloc((void**)&dbuf, bytes));

  p = (Body*)buf;

  randomizeBodies(buf, 6*nBodies); // Init pos / vel data

  CHECK_CUDA_ERROR(hipMemcpy(dbuf, buf, bytes, hipMemcpyHostToDevice));

  // CPU Implementation
  for (int iter = 1; iter <= nIters; iter++) {
    StartTimer();

    bodyForce(p, dt, nBodies); // compute interbody forces
    
    // #pragma omp parallel for
    for (int i = 0 ; i < nBodies; i++) { // integrate position
      p[i].x += p[i].vx*dt;
      p[i].y += p[i].vy*dt;
      p[i].z += p[i].vz*dt;
    }

    const double tElapsed = GetTimer() / 1000.0;
    if (iter > 1) { // First iter is warm up
      totalTime += tElapsed; 
    }
    printf("Iteration %d: %.3f seconds\n", iter, tElapsed);
    
    // Save 1st iteration's results for the correctness check
    if(iter == 1)
      memcpy(buf_1iter, buf, bytes);
  }
  avgTime = totalTime / (double)(nIters-1); 

  printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, 1e-9 * nBodies * nBodies / avgTime);
  extra_block = (nBodies%1024 != 0);
  //GPU Impkementation
  hipEventCreate(&startCuda);
  hipEventCreate(&stopCuda);

  hipEventRecord(startCuda, 0);
  for (int iter = 1; iter <= nIters; iter++) {
    bodyForce_gpu<<<(nBodies/1024)+extra_block, 1024>>>(dbuf, dt, nBodies); // compute interbody forces
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        break;
    }

    // Save 1st iter GPU results
    if(iter == 1){
      hipEventRecord(stopCuda, 0); // Stop timing before correctness check
      hipEventSynchronize(stopCuda);
      hipEventElapsedTime(&msecBeforeCheck, startCuda, stopCuda); // Time before correctness check
      CHECK_CUDA_ERROR(hipMemcpy(hbuf, dbuf, bytes, hipMemcpyDeviceToHost));
      // Compare CPU 1st iter results with GPU 1st iter results
      if(check_correctness(hbuf, buf_1iter, nBodies, &diff_counter)){
        printf("SUCCESS: Results are the same!");
      }
      else{
        printf("FAIL (differences = %d) : Results are diffesrent", diff_counter);
        goto cleanup;
      }
      hipEventRecord(startCuda, 0); // Restart timing after correctness check
    }
  }


  CHECK_CUDA_ERROR(hipMemcpy(hbuf, dbuf, bytes, hipMemcpyDeviceToHost));
  hipEventRecord(stopCuda, 0); // Final timing after the loop
  hipEventSynchronize(stopCuda);
  hipEventElapsedTime(&msecAfterCheck, startCuda, stopCuda); // Time after correctness check
  msec = msecBeforeCheck + msecAfterCheck;
  printf("GPU TIME: %f\n", msec);

  
  
//   for (int i = 0; i < nBodies; i++) {
//     // Compare positions and velocities between CPU and GPU
//     if (fabs(p[i].x - hbuf[i].x) > tolerance ||
//         fabs(p[i].y - hbuf[i].y) > tolerance ||
//         fabs(p[i].z - hbuf[i].z) > tolerance ||
//         fabs(p[i].vx - hbuf[i].vx) > tolerance ||
//         fabs(p[i].vy - hbuf[i].vy) > tolerance ||
//         fabs(p[i].vz - hbuf[i].vz) > tolerance) {
        
//         printf("Difference exceeds tolerance at index: %d\n", i);
//         printf("Position - CPU: (%.6f, %.6f, %.6f), GPU: (%.6f, %.6f, %.6f)\n",
//                p[i].x, p[i].y, p[i].z, hbuf[i].x, hbuf[i].y, hbuf[i].z);
//         printf("Velocity - CPU: (%.6f, %.6f, %.6f), GPU: (%.6f, %.6f, %.6f)\n",
//                p[i].vx, p[i].vy, p[i].vz, hbuf[i].vx, hbuf[i].vy, hbuf[i].vz);
//         goto cleanup;
//     }
// }
  
  
  
  
  
  cleanup:
  if (buf) free(buf);
  if (buf_1iter) free(buf_1iter);
  if (hbuf) free(hbuf);
  if (dbuf) hipFree(dbuf);

  hipDeviceSynchronize();
  e = hipGetLastError();
  if(e!=hipSuccess){
    printf("ERROR: %s, FILE: %s, LINE: %d\n", hipGetErrorString(e), __FILE__, __LINE__);
  }
  else{
    printf("hipGetLastError() == hipSuccess!\n");
  }

  hipDeviceReset();
  
  return 0;
}